#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""

#include <stdio.h>

#include "Image.h"
#include "PPM.h"

#include <cstdio>
#include <cassert>
#include <iostream>

#define _USE_MATH_DEFINES
#include <math.h>
#include <iomanip>

static void CheckCudaErrorAux(const char *, unsigned, const char *,
	hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

/**
* Check the return value of the CUDA runtime API call and exit
* the application if the call has failed.
*/
static void CheckCudaErrorAux(const char *file, unsigned line,
	const char *statement, hipError_t err) {
	if (err == hipSuccess)
		return;
	std::cerr << statement << " returned " << hipGetErrorString(err) << "("
		<< err << ") at " << file << ":" << line << std::endl;
	exit(1);
}

// useful defines
#define TILE_WIDTH 16
#define w (TILE_WIDTH + Mask_width - 1)
#define clamp(x) (min(max((x), 0.0), 1.0))

//Global variables
const int maskRows = 5;
const int maskColumns = 5;
const int maskRowsRadius = maskRows / 2;
const int maskColumnsRadius = maskColumns / 2;
__constant__ float deviceMaskData[maskRows * maskColumns];


__global__ void convolution(float *I, float *P,
	int channels, int width, int height) {

	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int depth = threadIdx.z;

	//Copia dato nella shared memory (Tiling)

	//Sincronizzati in atttedsa deglim altri
	__syncthreads();

	if (col < width && row < height && depth < channels) {
		//Evaluate convolution
		float pValue = 0;

		int startRow = row - maskRowsRadius;
		int startCol = col - maskColumnsRadius;

		for (int i = 0; i < maskRows; i++) {
			for (int j = 0; j < maskColumns; j++) {
				int currentRow = startRow + i;
				int currentColumn = startCol + j;

				pValue += I[(currentRow * width +  currentColumn) * channels + depth] * deviceMaskData[i * maskRows + j];
			}
		}

		//Salva il risultato dal registro alla global
		P[(row * width + col) * channels + depth] = pValue;
	}
}

// simple test to read/write PPM images, and process Image_t data
void test_images() {
	Image_t* inputImg = PPM_import("computer_programming.ppm");
	for (int i = 0; i < 300; i++) {
		Image_setPixel(inputImg, i, 100, 0, float(i) / 300);
		Image_setPixel(inputImg, i, 100, 1, float(i) / 300);
		Image_setPixel(inputImg, i, 100, 2, float(i) / 200);
	}
	PPM_export("test_output.ppm", inputImg);
	Image_t* newImg = PPM_import("test_output.ppm");
	inputImg = PPM_import("computer_programming.ppm");
	if (Image_is_same(inputImg, newImg))
		printf("Img uguali\n");
	else
		printf("Img diverse\n");
}

void constantFilter(float mask[])
{
	for (int x = 0; x < 5; x++)
	{
		for (int y = 0; y < 5; y++)
		{
			mask[x*5 + y] = (float)1 / (float)(5*5);
		}
	}
}

void identityFilter(float mask[])
{
	for (int x = 0; x < 5; x++)
	{
		for (int y = 0; y < 5; y++)
		{
			mask[x*5 + y] = 0;
		}
	}
	mask[2*5 + 2] = 1;
}

void gaussianFilter(float mask[])
{
	// means on X and Y are fixed to 0
	// correlation coefficient is fixed to 0
	// standard deviation is set to 1 (for both X and Y)
	float sigma = 1.0;
	float r, s = 2.0 * sigma * sigma;

	// sum is for normalization
	float sum = 0.0;

	// generate 5x5 mask values
	for (int x = -2; x <= 2; x++)
	{
		for (int y = -2; y <= 2; y++)
		{
			r = sqrt(x*x + y*y);
			mask[(x + 2)*5 + (y + 2)] = (exp(-(r*r) / s)) / (M_PI * s);
			sum += mask[(x + 2) * 5 + (y + 2)];
		}
	}

	// normalize the mask
	for (int i = 0; i < 5; ++i)
		for (int j = 0; j < 5; ++j)
			mask[i*5 + j] /= sum;

}

void printFilter(float mask[]) {
	for (int i = 0; i < 5; ++i)
	{
		for (int j = 0; j < 5; ++j)
			std::cout << mask[i*5 + j] << "\t";
		std::cout << std::endl;
	}
	std::cin.ignore();
}

int main() {

	int imageChannels;
	int imageWidth;
	int imageHeight;
	Image_t* inputImage;
	Image_t* outputImage;
	float *hostInputImageData;
	float *hostOutputImageData;
	float *deviceInputImageData;
	float *deviceOutputImageData;
	
	float hostMaskData[maskRows * maskColumns];
	gaussianFilter(hostMaskData);
	//printFilter(hostMaskData); // uncomment to check mask values

	inputImage = PPM_import("computer_programming.ppm");

	assert(maskRows == 5); /* mask height is fixed to 5 in this exercise */
	assert(maskColumns == 5); /* mask width is fixed to 5 in this exercise */

	imageWidth = Image_getWidth(inputImage);
	imageHeight = Image_getHeight(inputImage);
	imageChannels = Image_getChannels(inputImage);

	outputImage = Image_new(imageWidth, imageHeight, imageChannels);

	hostInputImageData = Image_getData(inputImage);
	hostOutputImageData = Image_getData(outputImage);

	// Allocate device buffers
	CUDA_CHECK_RETURN(
		hipMalloc((void **)&deviceInputImageData,
			sizeof(float) * imageWidth * imageHeight * imageChannels));

	CUDA_CHECK_RETURN(
		hipMalloc((void **)&deviceOutputImageData,
			sizeof(float) * imageWidth * imageHeight * imageChannels));


	//copy memory from host to device
	CUDA_CHECK_RETURN(
		hipMemcpyToSymbol(HIP_SYMBOL(deviceMaskData), hostMaskData, maskRows * maskColumns * sizeof(float)));
	CUDA_CHECK_RETURN(
		hipMemcpy(deviceInputImageData, hostInputImageData, sizeof(float) * imageWidth * imageHeight * imageChannels,
			hipMemcpyHostToDevice));

	//Evaluate block and thread number
	int requiredThread = (imageWidth + (maskColumnsRadius * 2)) * (imageHeight + (maskRowsRadius * 2)) * imageChannels;
	int numberThreadX = 16;
	int numberThreadY = 16;

	int numberBlockX = ceil((float)imageWidth /numberThreadX);
	int numberBlockY = ceil((float)imageHeight / numberThreadY);
		
	dim3 dimGrid(numberBlockX, numberBlockY); 
	dim3 dimBlock(numberThreadX, numberThreadY, 3); 
	convolution<<<dimGrid, dimBlock>>>(deviceInputImageData,
	 deviceOutputImageData, imageChannels, imageWidth, imageHeight);

	// Copy from device to host memory
	CUDA_CHECK_RETURN(
		hipMemcpy(hostOutputImageData, deviceOutputImageData, sizeof(float) * imageWidth * imageHeight * imageChannels,
			hipMemcpyDeviceToHost));

	PPM_export("processed_computer_programming.ppm", outputImage);

	// Free device memory
	//deviceMaskData memory doesn't need to be freed since it's a global variable
	hipFree(deviceInputImageData);
	hipFree(deviceOutputImageData);

	Image_delete(outputImage);
	Image_delete(inputImage);

	return 0;
}

